#include "hip/hip_runtime.h"
#include <stdio.h>
#include "getMin.h"


__device__ int getBlockMin(const float *values, int *index) {
	if(blockDim.x % 32 != 0)
		printf("Warning: block size isn't suited to reduction. getBlockMin may have error! #################\n");
	int tid = threadIdx.x;
	index[tid] = tid;
	__syncthreads();
	for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
		if (tid < offset) {
			if (values[index[tid + offset]] < values[index[tid]]) {
				index[tid] = index[tid + offset];
			}
		}
		__syncthreads();
	}
	return index[0];
}

__device__ void GetMinValueOriginal(real *pfValues)
{
	if(blockDim.x % 32 != 0)
		printf("Warning: block size isn't suited to reduction. getBlockMin may have error! #################\n");
	//Reduce by a factor of 2, and minimize step size
	for (int i = blockDim.x / 2; i > 0 ; i >>= 1) {
		int tid = threadIdx.x;
		if (tid < i)
			if (pfValues[tid + i] < pfValues[tid])
				pfValues[tid] = pfValues[tid + i];
        __syncthreads();
	}
}
