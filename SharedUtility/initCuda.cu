/*
 * initCuda.cpp
 *
 *  Created on: 10/12/2014
 *      Author: Zeyi Wen
 */

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>

using std::cout;
using std::cerr;
using std::endl;

/**
 * @brief: set the device to use
 */
void UseDevice(int deviceId, hipCtx_t &context)
{
    hipDevice_t device;
    hipDeviceProp_t prop;
    checkCudaErrors(hipGetDeviceProperties(&prop, deviceId));
    cout << "Using " << prop.name << "; device id is " << deviceId << endl;
    checkCudaErrors(hipSetDevice(deviceId));
    hipDeviceGet(&device, deviceId);
    hipCtxCreate(&context, hipDeviceMapHost, device);
    if(!prop.canMapHostMemory)
		fprintf(stderr, "Device %d cannot map host memory!\n", deviceId);
}

/**
 * @brief: get the device with maximum memory
 */
int GetMaxMemDevice(int count){
	int id = 0;
	unsigned int maxMem = 0;
	for(int i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        checkCudaErrors(hipGetDeviceProperties(&prop, i));
        if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
			//check memory size
			checkCudaErrors(hipSetDevice(i));
			size_t nFreeMem, nTotalMem;
			checkCudaErrors(hipMemGetInfo(&nFreeMem, &nTotalMem));  
			if(nFreeMem > maxMem){
				maxMem = nFreeMem;
				id = i;
			}
        }
    }
	return id;
}

/**
 * @brief: initialize CUDA device
 */

bool InitCUDA(hipCtx_t &context, char gpuType = 'T')
{
    int count;

    checkCudaErrors(hipGetDeviceCount(&count));
    if(count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

	//use the device with the largest available memory
	int bestId = GetMaxMemDevice(count);
    hipDeviceProp_t prop;
    checkCudaErrors(hipGetDeviceProperties(&prop, bestId));
	if(prop.name[0] == gpuType){
		UseDevice(bestId, context);
		return true;
	}

	//choose the device with the prefer name
    int i;
    for(i = 0; i < count; i++) {
        checkCudaErrors(hipGetDeviceProperties(&prop, i));
        if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
        	cout << prop.name << endl;
        	if(prop.name[0] == gpuType)
        	{//choose the prefer device
                UseDevice(i, context);
       			break;
        	}
        }
    }

    cout << i << " v.s. " << count << endl;
    if(i == count)
    {
        cout << "There is no device of \"" << gpuType << "\" series" << endl;
        UseDevice(0, context);
    }

    return true;
}

bool ReleaseCuda(hipCtx_t &context)
{
	cuCtxDetach(context);
	return true;
}
