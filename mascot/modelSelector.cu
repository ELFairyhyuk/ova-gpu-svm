
#include <hip/hip_runtime_api.h>
#include "modelSelector.h"
#include "../svm-shared/svmTrainer.h"
#include "../svm-shared/HessianIO/baseHessian.h"
#include "../svm-shared/HessianIO/parAccessor.h"
#include "../svm-shared/HessianIO/seqAccessor.h"
#include "../svm-shared/storageManager.h"
#include "../SharedUtility/DataType.h"

/**
 * @brief: search the best pair of parameters
 */
bool CModelSelector::GridSearch(const Grid &SGrid, vector<vector<real> > &v_vDocVector, vector<int> &vnLabel)
{
	bool bReturn = false;

	const vector<real> &vfGamma = SGrid.vfGamma;
	const vector<real> &vfC = SGrid.vfC;

	int nNumofSample = v_vDocVector.size();
	int *pnPredictedLabel = new int[nNumofSample];
	int nNumofFold = 10;//10 means 10-fold cross

	DeviceHessian::m_nTotalNumofInstance = v_vDocVector.size();
	CLRUCache cacheStrategy(v_vDocVector.size());
	cout << "using " << cacheStrategy.GetStrategy() << " caching strategy"<< endl;

	ofstream confusion;
	confusion.open("matrix.txt", std::ofstream::out | std::ofstream::app);

	for(uint j = 0; j < vfGamma.size(); j++)
	{
		CRBFKernel rbf(vfGamma[j]);//ignore
		DeviceHessian hessianIOOps(&rbf);
		//initial Hessian accessor
		SeqAccessor accessor;
		accessor.m_nTotalNumofInstance = DeviceHessian::m_nTotalNumofInstance;
		accessor.SetInvolveData(0, accessor.m_nTotalNumofInstance - 1, -1, -1);

		hessianIOOps.SetAccessor(&accessor);

		CSVMPredictor svmPredicter(&hessianIOOps);

		hessianIOOps.PrecomputeKernelMatrix(v_vDocVector, &hessianIOOps);

		//start n-fold-cross-validation, by changing C for SVM
		for(uint k = 0; k < vfC.size(); k++)
		{
			CSMOSolver s(&hessianIOOps, &cacheStrategy);

			CSVMTrainer svmTrainer(&s);
			m_pTrainer = &svmTrainer;
			m_pPredictor = &svmPredicter;

			memset(pnPredictedLabel, 0, sizeof(int) * nNumofSample);
			gfNCost = vfC[k];
			gfPCost = vfC[k];
			gfGamma = vfGamma[j];
			ofstream writeOut(OUTPUT_FILE, ios::app | ios::out);
			writeOut << "Gamma=" << vfGamma[j] << "; Cost=" << vfC[k] << endl;

			timespec timeValidS, timeValidE;
			clock_gettime(CLOCK_REALTIME, &timeValidS);
			bool bCrossValidation = CrossValidation(nNumofFold, vnLabel, pnPredictedLabel);
			if(bCrossValidation == false)
			{
				cerr << "can't have valid result in N_fold_cross_validation" << endl;
				continue;
			}
			clock_gettime(CLOCK_REALTIME, &timeValidE);
			long lCrossValidationTime = ((timeValidE.tv_sec - timeValidS.tv_sec) * 1e9 + (timeValidE.tv_nsec - timeValidS.tv_nsec));
			writeOut.close();
			//output n-fold-cross-validation result
			OutputResult(confusion, vnLabel, pnPredictedLabel, nNumofSample);

			cout << "total time: " << (double)lCrossValidationTime / 1000000 << "ms" << endl;
		}//end varying C
		//release pinned memory
		hipHostFree(DeviceHessian::m_pfHessianRowsInHostMem);
		delete[] DeviceHessian::m_pfHessianDiag;
	}//end varying gamma

	delete[] pnPredictedLabel;

	return bReturn;
}

/*
 * @brief: n fold cross validation
 * @param: nFold: the number of fold for the cross validation
 */
bool CModelSelector::CrossValidation(const int &nFold, vector<int> &vnLabel, int *&pnPredictedLabel)
{
	bool bReturn = true;

	int nTotalNumofSamples = vnLabel.size();
	//get labels of data
	int *pnLabelAll = new int[nTotalNumofSamples];
	for(int l = 0; l < nTotalNumofSamples; l++)
	{
		if(vnLabel[l] != 1 && vnLabel[l] != -1)
		{
			cerr << "error label (valid label is -1 or 1): " << vnLabel[l] << endl;
			cerr << "multi-class problems are not supported" << endl;
			exit(0);
		}
		pnLabelAll[l] = vnLabel[l];
	}

	//check input parameters
	if((nFold < 1) || nTotalNumofSamples < nFold)
	{
		cerr << "error in cross validation: invalid parameters" << endl;
		exit(0);
	}

	//divide the training samples in to n folds. note that the last fold may be larger than other folds.
	int nSizeofFold = 0;
	nSizeofFold = nTotalNumofSamples / nFold;
	int *pnFoldStart = new int[nFold];
	//Initialise the first fold
	pnFoldStart[0] = 0;
	//for the case that there is only one fold
	if(nFold == 1)
	{
		pnFoldStart[1] = 0;
	}
	//start counting the size of each fold
	for(int i = 1; i < nFold; i++)
	{
		pnFoldStart[i] = pnFoldStart[i - 1] + nSizeofFold;
	}

	//during n-fold cross validation, training samples are divided into at most 2 parts
	int *pnSizeofParts = new int[2];

	/* allocate GPU device memory */
	//set default value at
	real *pfAlphaAll;
	real *pfYiGValueAll;
	pfAlphaAll = new real[nTotalNumofSamples];
	pfYiGValueAll = new real[nTotalNumofSamples];
	for(int i = 0; i < nTotalNumofSamples; i++)
	{
		//initially, the values of alphas are 0s
		pfAlphaAll[i] = 0;
		//GValue is -y_i, as all alphas are 0s. YiGValue is always -1
		pfYiGValueAll[i] = -pnLabelAll[i];
	}

	/* start n-fold-cross-validation */
	//allocate GPU memory for part of samples that are used to perform training.
	real *pfDevAlphaSubset;
	real *pfDevYiGValueSubset;
	int *pnDevLabelSubset;

	real *pfPredictionResult = new real[nTotalNumofSamples];
	for(int i = 0; i < nFold; i++)
	{
		/**************** training *******************/
		//first continual part of sample data
		if(i != 0)
		{
			pnSizeofParts[0] = pnFoldStart[i];
		}
		else
		{
			pnSizeofParts[0] = 0;
		}

		//second continual
		if(i != nFold - 1 || nFold == 1)//nFold == 1 is  for special case, where all samples are for training and testing
		{
			pnSizeofParts[1] = nTotalNumofSamples - pnFoldStart[i + 1];
		}
		else
		{
			pnSizeofParts[1] = 0;
		}

		//get size of training samples
		int nNumofTrainingSamples = 0;
		nNumofTrainingSamples = pnSizeofParts[0] + pnSizeofParts[1];

		//in n-fold-cross validation, the first (n -1) parts have the same size, so we can reuse memory
		if(i == 0 || (i == nFold - 1))
		{
			checkCudaErrors(hipMalloc((void**)&pfDevAlphaSubset, sizeof(real) * nNumofTrainingSamples));
			checkCudaErrors(hipMalloc((void**)&pfDevYiGValueSubset, sizeof(real) * nNumofTrainingSamples));
			checkCudaErrors(hipMalloc((void**)&pnDevLabelSubset, sizeof(int) * nNumofTrainingSamples));
		}
		//set GPU memory
		checkCudaErrors(hipMemset(pfDevAlphaSubset, 0, sizeof(real) * nNumofTrainingSamples));
		checkCudaErrors(hipMemset(pfDevYiGValueSubset, -1, sizeof(real) * nNumofTrainingSamples));
		checkCudaErrors(hipMemset(pnDevLabelSubset, 0, sizeof(int) * nNumofTrainingSamples));
		//copy training information to GPU for current training
		checkCudaErrors(hipMemcpy(pfDevAlphaSubset, pfAlphaAll,
								   sizeof(real) * pnSizeofParts[0], hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(pfDevYiGValueSubset, pfYiGValueAll,
								   sizeof(real) * pnSizeofParts[0], hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(pnDevLabelSubset, pnLabelAll,
								   sizeof(int) * pnSizeofParts[0], hipMemcpyHostToDevice));
		//part two
		if(pnSizeofParts[1] != 0)
		{
			checkCudaErrors(hipMemcpy(pfDevAlphaSubset + pnSizeofParts[0], pfAlphaAll + pnFoldStart[i + 1],
									   sizeof(real) * pnSizeofParts[1], hipMemcpyHostToDevice));
			checkCudaErrors(hipMemcpy(pfDevYiGValueSubset + pnSizeofParts[0], pfYiGValueAll + pnFoldStart[i + 1],
									   sizeof(real) * pnSizeofParts[1], hipMemcpyHostToDevice));
			checkCudaErrors(hipMemcpy(pnDevLabelSubset + pnSizeofParts[0], pnLabelAll + pnFoldStart[i + 1],
									   sizeof(int) * pnSizeofParts[1], hipMemcpyHostToDevice));
		}

		/************** train SVM model **************/
		int nSampleStart1, nSampleEnd1, nSampleStart2, nSampleEnd2;
		if(pnSizeofParts[0] != 0)
		{
			nSampleStart1 = 0;
			nSampleEnd1 = pnSizeofParts[0] - 1;
		}
		else
		{
			nSampleStart1 = -1;
			nSampleEnd1 = -1;
		}
		if(pnSizeofParts[1] != 0)
		{
			nSampleStart2 = pnFoldStart[i + 1];
			nSampleEnd2 = nTotalNumofSamples - 1;
		}
		else
		{
			nSampleStart2 = -1;
			nSampleEnd2 = -1;
		}

		cout << "training the " << i + 1 << "th classifier";
		cout.flush();

		svm_model model;
		//set data involved in training
		m_pTrainer->SetInvolveTrainingData(nSampleStart1, nSampleEnd1, nSampleStart2, nSampleEnd2);
		bool bTrain = m_pTrainer->TrainModel(model, pfDevYiGValueSubset, pfDevAlphaSubset, pnDevLabelSubset, nNumofTrainingSamples, NULL);
		if(bTrain == false)
		{
			cerr << "can't find an optimal classifier" << endl;
			bReturn = false;
			break;
		}

		/******************** prediction *******************/
		//get the size of a fold for testing
		int nNumofTestingSample = 0;
		if(i != nFold - 1)
			nNumofTestingSample = pnFoldStart[i + 1] - pnFoldStart[i];
		else
			nNumofTestingSample = nTotalNumofSamples - pnFoldStart[i];

		//get testing sample id
		int *pnTestSampleId;
		if(i == 0 || (i == nFold - 1))
		{
			pnTestSampleId = new int[nNumofTestingSample];
		}
		for(int j = 0; j < nNumofTestingSample; j++)
		{
			pnTestSampleId[j] = pnFoldStart[i] + j;
		}

		timespec timeClassificationS, timeClassificationE;
		clock_gettime(CLOCK_REALTIME, &timeClassificationS);

		cout << "performing classification...";
		//set data involve in prediction
		m_pPredictor->SetInvolvePredictionData(pnTestSampleId[0], pnTestSampleId[nNumofTestingSample - 1]);
		//perform prediction
		real *pfPartialPredictionResult;
		pfPartialPredictionResult = m_pPredictor->Predict(&model, pnTestSampleId, nNumofTestingSample);
		cout << " Done"<< endl;
		clock_gettime(CLOCK_REALTIME, &timeClassificationE);

		//copy partial result to global result
		for(int j = 0; j < nNumofTestingSample; j++)
		{
			pfPredictionResult[pnFoldStart[i] + j] = pfPartialPredictionResult[j];
		}

		//for comparing the results of the other gpu svm
		int nCorrect = 0;
		for(int j = 0; j < nNumofTestingSample; j++)
		{
			if(pfPartialPredictionResult[j] > 0 && pnLabelAll[pnFoldStart[i] + j] > 0)
				nCorrect++;
			else if(pfPartialPredictionResult[j] < 0 && pnLabelAll[pnFoldStart[i] + j] < 0)
				nCorrect++;
		}
		cout << "accuracy in this fold: " << (float)nCorrect/nNumofTestingSample << endl;

		delete[] pfPartialPredictionResult; //as memory is allocated during prediction
		//release memory, in the first (nFold - 2) iterations, the space of pnTestSampleId can be reused
		if(i >= (nFold - 2))
		{
			delete[] pnTestSampleId;
		}
		DestroySVMModel(model);
	}

	//calculate classification label
	for(int i = 0; i < nTotalNumofSamples; i++)
	{
		if(pfPredictionResult[i] > 0)
		{
			pnPredictedLabel[i] = 1;
		}
		else
		{
			pnPredictedLabel[i] = -1;
		}
	}

	checkCudaErrors(hipFree(pfDevAlphaSubset));
	checkCudaErrors(hipFree(pnDevLabelSubset));
	checkCudaErrors(hipFree(pfDevYiGValueSubset));

	delete[] pfAlphaAll;
	delete[] pfYiGValueAll;

	delete[] pfPredictionResult;
	delete[] pnSizeofParts;

	delete[] pnFoldStart;
	return bReturn;
}

/*
 * @brief: output prediction result (e.g., accuracy, recall, precision etc.)
 * @param: pnOriginalLabel: label of training samples
 * @param: pnPredictedLabel: label assigned by SVM
 */
bool CModelSelector::OutputResult(ofstream &confusion, vector<int> &pnOriginalLabel, int *pnPredictedLabel, int nSizeofSample)
{
	bool bReturn = false;
	int nCorrect = 0, nTrueP = 0, nFalseP = 0, nFalseN = 0, nTrueN = 0;
	for(int i = 0; i < nSizeofSample; i++)
    {
		if((pnPredictedLabel[i] == 1 && pnOriginalLabel[i] == 1) || (pnPredictedLabel[i] == -1 && pnOriginalLabel[i] == -1))
		{
			nCorrect++;
		}

		if(pnPredictedLabel[i] == 1 && pnOriginalLabel[i] == 1)
		{
			nTrueP++;
		}
		else if(pnPredictedLabel[i] == -1 && pnOriginalLabel[i] == 1)
		{
			nFalseN++;
		}
		else if(pnPredictedLabel[i] == 1 && pnOriginalLabel[i] == -1)
		{
			nFalseP++;
		}
		else if(pnPredictedLabel[i] == -1 && pnOriginalLabel[i] == -1)
		{
			nTrueN++;
		}
		else
		{
			cout << "error in output result: " << pnPredictedLabel[i] << " and " << pnOriginalLabel[i]
			     << " are not equal to +1 or -1" << endl;
		}
    }

	double dPrecision = (double)nTrueP / (nTrueP + nFalseP);
	double dRecall = (double)nTrueP / (nFalseN + nTrueP);

	confusion << "TP: " << nTrueP << " FN: " << nFalseN << ";\t";
	confusion << "FP: " << nFalseP<< " TN: " << nTrueN << endl;

	cout << "Accuracy: " << (double)nCorrect / nSizeofSample << " precision@pos: " << dPrecision << " recall@pos: " << dRecall << endl;
	cout << "Precision@neg: " << (double)nTrueN/(nTrueN + nFalseN) << " recall@neg: " << (double)nTrueN/(nTrueN + nFalseP) << endl;

/*	ofstream writeOut(OUTPUT_FILE, ios::app | ios::out);
	writeOut << "true positive: " << nTrueP << "; true negative: " << nTrueN
			 << "; false positive: " << nFalseP << "; false negative: " << nFalseN << endl;
	writeOut << (double)nCorrect / nSizeofSample << " precision: " << dPrecision
				<< " recall: " << dRecall << endl;
	writeOut.close();
*/
	return bReturn;
}

/*
 * @brief: destroy svm model
 */
bool CModelSelector::DestroySVMModel(svm_model &model)
{
	bool bReturn = true;

	delete[] model.label;
	delete[] model.pnIndexofSV;
	delete[] model.rho;
	delete[] model.sv_coef[0];
	delete[] model.sv_coef[1];
	delete[] model.sv_coef[2];
	delete[] model.sv_coef;

	return bReturn;
}
