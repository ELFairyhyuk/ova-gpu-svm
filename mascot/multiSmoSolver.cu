#include "hip/hip_runtime.h"
//
// Created by ss on 16-12-14.
//

#include <cfloat>
#include <sys/time.h>
#include <thrust/sort.h>
#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include "multiSmoSolver.h"
#include "trainClassifier.h"
#include "../svm-shared/constant.h"
#include "../svm-shared/smoGPUHelper.h"
#include "../svm-shared/HessianIO/deviceHessianOnFly.h"
#include "../svm-shared/Cache/subHessianCalculator.h"
#include "../SharedUtility/getMin.h"
#include "../SharedUtility/Timer.h"
#include "../SharedUtility/powerOfTwo.h"
#include "../SharedUtility/CudaMacro.h"
#include "multiPredictor.h"

void MultiSmoSolver::solve() {
    int nrClass = problem.getNumOfClasses();

    if (model.vC.size() == 0) {//initialize C for all the binary classes
        model.vC = vector<real>(nrClass * (nrClass - 1) / 2, param.C);
    }

    //train nrClass*(nrClass-1)/2 binary models
    int k = 0;
    vector<int> prob_start(problem.start);
	
	std::map<int , int>indexMap;
	indexMap.clear();
    
    for (int i = 0; i < nrClass; ++i) {
        int ci = problem.count[i];
        for (int j = i + 1; j < nrClass; ++j) {
            printf("training classifier with label %d and %d\n", i, j);
            SvmProblem subProblem = problem.getSubProblem(i, j);

            //determine the size of working set
            workingSetSize = 1024;
            int minSize = min(subProblem.count[0], subProblem.count[1]);
            if (minSize< workingSetSize) {
                workingSetSize = floorPow2(minSize);
            }
            q = workingSetSize / 2;
            printf("q = %d, working set size = %d\n", q, workingSetSize);

            //should be called after workingSetSize is initialized
            init4Training(subProblem);

            //convert binary sub-problem to csr matrix and copy it to device
            CSRMatrix subProblemMat(subProblem.v_vSamples, subProblem.getNumOfFeatures());
            subProblemMat.copy2Dev(devVal, devRowPtr, devColInd, devSelfDot);
            nnz = subProblemMat.getNnz();

            printf("#positive ins %d, #negative ins %d\n", subProblem.count[0], subProblem.count[1]);
            int totalIter = 0;
            TIMER_START(trainingTimer)
            //start binary svm training
            for (int l = 0;; ++l) {
                workingSetIndicator = vector<int>(subProblem.getNumOfSamples(), 0);
                if (l == 0) {
                    selectWorkingSetAndPreCompute(subProblem, workingSetSize / 2, model.vC[k]);
                } else {
                    for (int m = 0; m < workingSetSize - q; ++m) {
                        workingSetIndicator[workingSet[q + m]] = 1;
                    }
                    selectWorkingSetAndPreCompute(subProblem, q / 2, model.vC[k]);
                }
                TIMER_START(iterationTimer)
                localSMO << < 1, workingSetSize, workingSetSize * sizeof(float) * 3 + 2 * sizeof(float) >> >
                                                 (devLabel, devYiGValue, devAlpha, devAlphaDiff, devWorkingSet, workingSetSize, model.vC[k], devHessianMatrixCache, subProblem.getNumOfSamples());
                TIMER_STOP(iterationTimer)
                TIMER_START(updateGTimer)
                updateF << < gridSize, BLOCK_SIZE >> >
                                       (devYiGValue, devLabel, devWorkingSet, workingSetSize, devAlphaDiff, devHessianMatrixCache, subProblem.getNumOfSamples());
                TIMER_STOP(updateGTimer)
                float diff;
                checkCudaErrors(hipMemcpyFromSymbol(&diff, HIP_SYMBOL(devDiff), sizeof(real), 0, hipMemcpyDeviceToHost));
                if (l % 10 == 0)
                    printf(".");
                cout.flush();
                if (diff < EPS) {
                    printf("\nup + low = %f\n", diff);
                    break;
                }
            }
            TIMER_STOP(trainingTimer)
            printf("obj = %f\n", getObjValue(subProblem.getNumOfSamples()));
            subProblemMat.freeDev(devVal, devRowPtr, devColInd, devSelfDot);
            vector<int> svIndex;
            vector<real> coef;
            real rho;


            extractModel(subProblem, svIndex, coef, rho);
            model.getModelParam(subProblem, svIndex, coef, prob_start, ci, i, j);
            model.addBinaryModel(subProblem, svIndex, coef, rho, i, j, indexMap);
            k++;

            deinit4Training();
        }
    }


}

void MultiSmoSolver::init4Training(const SvmProblem &subProblem) {
    unsigned int trainingSize = subProblem.getNumOfSamples();

    workingSet = vector<int>(workingSetSize);
    checkCudaErrors(hipMalloc((void **) &devAlphaDiff, sizeof(real) * workingSetSize));
    checkCudaErrors(hipMalloc((void **) &devWorkingSet, sizeof(int) * workingSetSize));

    checkCudaErrors(hipMalloc((void **) &devAlpha, sizeof(real) * trainingSize));
    checkCudaErrors(hipMalloc((void **) &devYiGValue, sizeof(real) * trainingSize));
    checkCudaErrors(hipMalloc((void **) &devLabel, sizeof(int) * trainingSize));
    checkCudaErrors(hipMalloc((void **) &devWorkingSetIndicator, sizeof(int) * trainingSize));

    checkCudaErrors(hipMemset(devAlpha, 0, sizeof(real) * trainingSize));
    vector<real> negatedLabel(trainingSize);
    for (int i = 0; i < trainingSize; ++i) {
        negatedLabel[i] = -subProblem.v_nLabels[i];
    }
    checkCudaErrors(hipMemcpy(devYiGValue, negatedLabel.data(), sizeof(real) * trainingSize,
                               hipMemcpyHostToDevice));
    checkCudaErrors(
            hipMemcpy(devLabel, subProblem.v_nLabels.data(), sizeof(int) * trainingSize, hipMemcpyHostToDevice));

    InitSolver(trainingSize);//initialise base solver

    checkCudaErrors(hipMalloc((void **) &devHessianMatrixCache, sizeof(real) * workingSetSize * trainingSize));

    for (int j = 0; j < trainingSize; ++j) {
        hessianDiag[j] = 1;//assume using RBF kernel
    }
    checkCudaErrors(
            hipMemcpy(devHessianDiag, hessianDiag, sizeof(real) * trainingSize, hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void **) &devFValue4Sort, sizeof(real) * trainingSize));
    checkCudaErrors(hipMalloc((void **) &devIdx4Sort, sizeof(int) * trainingSize));

}

void MultiSmoSolver::deinit4Training() {
    checkCudaErrors(hipFree(devAlphaDiff));
    checkCudaErrors(hipFree(devWorkingSet));

    checkCudaErrors(hipFree(devAlpha));
    checkCudaErrors(hipFree(devYiGValue));
    checkCudaErrors(hipFree(devLabel));
    checkCudaErrors(hipFree(devWorkingSetIndicator));

    DeInitSolver();

    checkCudaErrors(hipFree(devHessianMatrixCache));
    checkCudaErrors(hipFree(devFValue4Sort));
    checkCudaErrors(hipFree(devIdx4Sort));
}

void MultiSmoSolver::extractModel(const SvmProblem &subProblem, vector<int> &svIndex, vector<real> &coef,
                                  real &rho) const {
    const unsigned int trainingSize = subProblem.getNumOfSamples();
    vector<real> alpha(trainingSize);
    const vector<int> &label = subProblem.v_nLabels;
    checkCudaErrors(hipMemcpy(alpha.data(), devAlpha, sizeof(real) * trainingSize, hipMemcpyDeviceToHost));
    for (int i = 0; i < trainingSize; ++i) {
        if (alpha[i] != 0) {
            coef.push_back(label[i] * alpha[i]);
            svIndex.push_back(i);

        }
    }
    checkCudaErrors(hipMemcpyFromSymbol(&rho, HIP_SYMBOL(devRho), sizeof(real), 0, hipMemcpyDeviceToHost));
    printf("# of SV %lu\nbias = %f\n", svIndex.size(), rho);
}

MultiSmoSolver::MultiSmoSolver(const SvmProblem &problem, SvmModel &model, const SVMParam &param) :
        problem(problem), model(model), param(param) {
}

MultiSmoSolver::~MultiSmoSolver() {
}

void
MultiSmoSolver::selectWorkingSetAndPreCompute(const SvmProblem &subProblem, uint numOfSelectPairs, real penaltyC) {
    uint numOfSamples = subProblem.getNumOfSamples();
    uint oldSize = workingSetSize / 2 - numOfSelectPairs;
    TIMER_START(selectTimer)
    thrust::device_ptr<float> valuePointer = thrust::device_pointer_cast(devFValue4Sort);
    thrust::device_ptr<int> indexPointer = thrust::device_pointer_cast(devIdx4Sort);
    vector<int> oldWorkingSet = workingSet;

    checkCudaErrors(hipMemcpy(devWorkingSetIndicator, workingSetIndicator.data(), sizeof(int) * numOfSamples,
                               hipMemcpyHostToDevice));

    //get q most violation pairs
    getFUpValues << < gridSize, BLOCK_SIZE >> >
                                (devYiGValue, devAlpha, devLabel, numOfSamples, penaltyC, devFValue4Sort, devIdx4Sort, devWorkingSetIndicator);
    thrust::sort_by_key(valuePointer, valuePointer + numOfSamples, indexPointer, thrust::greater<float>());
    checkCudaErrors(hipMemcpy(workingSet.data() + oldSize * 2, devIdx4Sort, sizeof(int) * numOfSelectPairs,
                               hipMemcpyDeviceToHost));
    for (int i = 0; i < numOfSelectPairs; ++i) {
        workingSetIndicator[workingSet[oldSize * 2 + i]] = 1;
    }
    checkCudaErrors(hipMemcpy(devWorkingSetIndicator, workingSetIndicator.data(), sizeof(int) * numOfSamples,
                               hipMemcpyHostToDevice));
    getFLowValues << < gridSize, BLOCK_SIZE >> >
                                 (devYiGValue, devAlpha, devLabel, numOfSamples, penaltyC, devFValue4Sort, devIdx4Sort, devWorkingSetIndicator);
    thrust::sort_by_key(valuePointer, valuePointer + numOfSamples, indexPointer, thrust::greater<float>());
    checkCudaErrors(
            hipMemcpy(workingSet.data() + oldSize * 2 + numOfSelectPairs, devIdx4Sort, sizeof(int) * numOfSelectPairs,
                       hipMemcpyDeviceToHost));

    //get pairs from last working set
    for (int i = 0; i < oldSize * 2; ++i) {
        workingSet[i] = oldWorkingSet[numOfSelectPairs * 2 + i];
    }
    checkCudaErrors(hipMemcpy(devWorkingSet, workingSet.data(), sizeof(int) * workingSetSize, hipMemcpyHostToDevice));
    TIMER_STOP(selectTimer)

    //move old kernel values to get space
    checkCudaErrors(hipMemcpy(devHessianMatrixCache,
                               devHessianMatrixCache + numOfSamples * numOfSelectPairs * 2,
                               sizeof(real) * numOfSamples * oldSize * 2,
                               hipMemcpyDeviceToDevice));
    vector<vector<KeyValue> > computeSamples;
    for (int i = 0; i < numOfSelectPairs * 2; ++i) {
        computeSamples.push_back(subProblem.v_vSamples[workingSet[oldSize * 2 + i]]);
    }
    TIMER_START(preComputeTimer)
    //preCompute kernel values of new selected instances
    hipsparseHandle_t handle;
    hipsparseMatDescr_t descr;
    CSRMatrix workingSetMat(computeSamples, subProblem.getNumOfFeatures());
    real * devWSVal;
    int *devWSColInd;
    int *devWSRowPtr;
    real * devWSSelfDot;
    workingSetMat.copy2Dev(devWSVal, devWSRowPtr, devWSColInd, devWSSelfDot);
    SubHessianCalculator::prepareCSRContext(handle, descr);
    CSRMatrix::CSRmm2Dense(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE, numOfSelectPairs * 2,
                           numOfSamples, subProblem.getNumOfFeatures(), descr,
                           workingSetMat.getNnz(), devWSVal, devWSRowPtr, devWSColInd, descr, nnz, devVal, devRowPtr,
                           devColInd, devHessianMatrixCache + numOfSamples * oldSize * 2);
    RBFKernel << < Ceil(numOfSelectPairs * 2 * numOfSamples, BLOCK_SIZE), BLOCK_SIZE >> > (devWSSelfDot, devSelfDot,
            devHessianMatrixCache + numOfSamples * oldSize * 2, numOfSelectPairs * 2, numOfSamples, param.gamma);
    SubHessianCalculator::releaseCSRContext(handle, descr);
    workingSetMat.freeDev(devWSVal, devWSRowPtr, devWSColInd, devWSSelfDot);
    TIMER_STOP(preComputeTimer)
}

real MultiSmoSolver::getObjValue(int numOfSamples) const {
    //the function should be called before deinit4Training
    vector<real> f(numOfSamples);
    vector<real> alpha(numOfSamples);
    vector<int> y(numOfSamples);
    hipMemcpy(f.data(), devYiGValue, sizeof(real) * numOfSamples, hipMemcpyDeviceToHost);
    hipMemcpy(alpha.data(), devAlpha, sizeof(real) * numOfSamples, hipMemcpyDeviceToHost);
    hipMemcpy(y.data(), devLabel, sizeof(int) * numOfSamples, hipMemcpyDeviceToHost);
    real obj = 0;
    for (int i = 0; i < numOfSamples; ++i) {
        obj -= alpha[i];
    }
    for (int i = 0; i < numOfSamples; ++i) {
            obj += 0.5 * alpha[i] * y[i] * (f[i] + y[i]);
    }
    return obj;
}

