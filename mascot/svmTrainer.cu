/*
 * @brief: this file contains the definition of svm trainer class
 * Created on: May 24, 2012
 * Author: Zeyi Wen
 * Copyright @DBGroup University of Melbourne
 */

#include "../svm-shared/svmTrainer.h"
#include "time.h"
#include "../svm-shared/gpu_global_utility.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <sys/time.h>

long nTimeOfLoop = 0;
long nTimeOfPrep = 0;

/*
 * @brief: train svm model. Training data is consisted of two parts for n-fold cross validation scenario;
 * @param: model: the svm model of the training
 * @param: pfDevYiFValueSubset: gradient of sub set of samples (training samples are sub set of the whole samples)
 */
int nInterater = 0;
bool CSVMTrainer::TrainModel(svm_model &model, real *pfDevYiFValueSubset, real *pfDevAlphaSubset,
							 int *pnDevLabelSubset, int nNumofInstance, real *pfP){
	bool bReturn = true;

	assert(nNumofInstance > 0  && pfDevAlphaSubset != NULL &&
		   pfDevYiFValueSubset != NULL && pnDevLabelSubset != NULL);

	/*************** prepare to perform training *************/
	TrainStarting(nNumofInstance, nNumofInstance, pfDevYiFValueSubset, pfDevAlphaSubset, pnDevLabelSubset);

	//start training process
	int nIter = 0;
	int nMaxIter = (nNumofInstance > INT_MAX / ITERATION_FACTOR ? INT_MAX : ITERATION_FACTOR * nNumofInstance) * 4;
	int nSelectFirstSample = -1;
	int nSelectSecondeSample = -1;

	timespec timeLoopS, timeLoopE;
	clock_gettime(CLOCK_REALTIME, &timeLoopS);

	while(nIter < nMaxIter)
	{
		int nEnd = m_pSMOSolver->Iterate(pfDevYiFValueSubset, pfDevAlphaSubset, pnDevLabelSubset, nNumofInstance);

		if(nEnd == 1)
		{
			cout << " Done" << endl;
			break;
		}
		if(nIter % 1000 == 0 && nIter != 0)
		{
            cout << ".";
			cout.flush();
		}
		nIter++;

	}
	clock_gettime(CLOCK_REALTIME, &timeLoopE);
	long lTempLoop = ((timeLoopE.tv_sec - timeLoopS.tv_sec) * 1e9 + (timeLoopE.tv_nsec - timeLoopS.tv_nsec));
	if(lTempLoop > 0)
		nTimeOfLoop += lTempLoop;
	else
		cout << "loop timer error" << endl;

	TrainEnding(nIter, nNumofInstance, nNumofInstance, model,
				pnDevLabelSubset, pfDevAlphaSubset, pfDevYiFValueSubset, pfP);

	//can't find a optimal classifier
	if(nIter == nMaxIter)
	{
		bReturn = false;
	}
	return bReturn;
}
