#include "hip/hip_runtime.h"
/*
 * subHessianCalculater.cu
 *
 *  Created on: 10 Jan 2017
 *      Author: Zeyi Wen
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <sys/time.h>
#include "subHessianCalculator.h"
#include "../constant.h"
#include "../../SharedUtility/Timer.h"
#include "../../SharedUtility/KeyValue.h"
#include "../../SharedUtility/CudaMacro.h"

__global__ void RBFKernel(const real *selfDot0, const real *selfDot1,
                          real *dotProduct, int n, int m,
                          float gamma) {
    const int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int i = idx / m;
    int j = idx % m;
    if (idx < n * m) {
        dotProduct[idx] = expf(-(selfDot0[i] + selfDot1[j] - dotProduct[idx] * 2) * gamma);
    }
}

/**
 * @brief: create handle and descr for CSR matrix operations
 */
void SubHessianCalculator::prepareCSRContext(hipsparseHandle_t &handle, hipsparseMatDescr_t &descr){
    hipsparseCreate(&handle);
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
}

/**
 * @brief: release handle and descr
 */
void SubHessianCalculator::releaseCSRContext(hipsparseHandle_t &handle, hipsparseMatDescr_t &descr){
    hipsparseDestroy(handle);
    hipsparseDestroyMatDescr(descr);
}

/**
 * @brief: compute a sub/whole kernel matrix
 * @param: n is the number of rows of matrix0; m is the number of rows of matrix1; k is the dimension.
 */
void SubHessianCalculator::computeSubHessianMatrix(hipsparseHandle_t handle, hipsparseMatDescr_t descr,
									   CSRMatrix &csrMatrix0, int n, CSRMatrix &csrMatrix1, int m, int k,
									   real *devC, const SVMParam &param){
	real *devVal0;
	int *devRowPtr0, *devColInd0;
	csrMatrix0.copy2Dev(devVal0, devRowPtr0, devColInd0);
	real *devSelfDot0;
	int nnz0 = csrMatrix0.getNnz();
	checkCudaErrors(hipMalloc((void **) &devSelfDot0, sizeof(real) * n));
	checkCudaErrors(hipMemcpy(devSelfDot0, csrMatrix0.getCSRValSelfDot(), sizeof(real) * n, hipMemcpyHostToDevice));

	//initialize parameters of matrix1
	int nnz1 = nnz0;
	real *devVal1 = devVal0;
	int *devRowPtr1 = devRowPtr0, *devColInd1 = devColInd0;
	real *devSelfDot1 = devSelfDot0;
	if(&csrMatrix1 != &csrMatrix0){//compare two addresses
		csrMatrix1.copy2Dev(devVal1, devRowPtr1, devColInd1);
		nnz1 = csrMatrix1.getNnz();
		checkCudaErrors(hipMalloc((void **) &devSelfDot1, sizeof(real) * m));
		checkCudaErrors(hipMemcpy(devSelfDot1, csrMatrix1.getCSRValSelfDot(), sizeof(real) * m, hipMemcpyHostToDevice));
	}
	CSRMatrix::CSRmm2Dense(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE, n, m, k, descr,
	                       nnz0, devVal0, devRowPtr0, devColInd0, descr, nnz1, devVal1, devRowPtr1, devColInd1, devC);
	RBFKernel << < Ceil(n * m, BLOCK_SIZE), BLOCK_SIZE >> > (devSelfDot0, devSelfDot1, devC, n, m, param.gamma);

	checkCudaErrors(hipFree(devSelfDot0));
    csrMatrix0.freeDev(devVal0, devRowPtr0, devColInd0);
    if(&csrMatrix1 != &csrMatrix0){
    	checkCudaErrors(hipFree(devSelfDot1));
    	csrMatrix1.freeDev(devVal1, devRowPtr1, devColInd1);
    }
}

void SubHessianCalculator::preComputeSharedCache(vector<real*> &hostSharedCache, const SvmProblem &problem,
                                                 const SVMParam &param) {
    hipsparseHandle_t handle;
    hipsparseMatDescr_t descr;
    prepareCSRContext(handle, descr);

    for (int i = 0; i < problem.getNumOfClasses(); ++i) {
        printf("pre-compute shared cache %d\n", i);
        vector<vector<KeyValue> > oneClass = problem.getOneClassSamples(i);
        int n = oneClass.size();
        int k = problem.getNumOfFeatures();
        CSRMatrix csrMatrix(oneClass, k);
        real *devC;
        checkCudaErrors(hipMalloc((void **) &devC, sizeof(real) * n * n));//this can be moved out of for-loop by reusing the memory.
        computeSubHessianMatrix(handle, descr, csrMatrix, n, csrMatrix, n, k, devC, param);

        checkCudaErrors(hipMemcpy(hostSharedCache[i], devC, sizeof(real) * n * n, hipMemcpyDeviceToHost));
        checkCudaErrors(hipFree(devC));
    }

    releaseCSRContext(handle, descr);
}

void SubHessianCalculator::preComputeUniqueCache(int i, int j, const SvmProblem &subProblem,
		    	vector<real*> &devUniqueCache, vector<size_t> &sizeOfEachRowInUniqueCache,
				vector<int> &numOfElementEachRowInUniqueCache, const SVMParam &param) {
    printf("pre-compute unique cache....");
    hipsparseHandle_t handle;
    hipsparseMatDescr_t descr;
    prepareCSRContext(handle, descr);

    int n = subProblem.count[0];
    int m = subProblem.count[1];
    int k = subProblem.getNumOfFeatures();
    vector<vector<KeyValue> > samples0(subProblem.v_vSamples.begin(), subProblem.v_vSamples.begin() + n);
    vector<vector<KeyValue> > samples1(subProblem.v_vSamples.begin() + n, subProblem.v_vSamples.begin() + n + m);
    CSRMatrix csrMatrix0(samples0, k);
    CSRMatrix csrMatrix1(samples1, k);
    real *devC;
    checkCudaErrors(hipMalloc((void **) &devC, sizeof(real) * n * m));
    computeSubHessianMatrix(handle, descr, csrMatrix0, n, csrMatrix1, m, k, devC, param);

    checkCudaErrors(hipMemcpy2D(devUniqueCache[0], sizeOfEachRowInUniqueCache[0], devC,
                                 m * sizeof(real), m * sizeof(real), n, hipMemcpyDeviceToDevice));

    //compute another sub kernel matrix by transposition
    float const alpha(1.0);
    float const beta(0.0);
    hipblasHandle_t handle2;
    hipblasCreate(&handle2);
    hipblasSgeam(handle2, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, &alpha, devC, m, &beta, devC, n, devUniqueCache[1],
                numOfElementEachRowInUniqueCache[1]);
    hipblasDestroy(handle2);

    checkCudaErrors(hipFree(devC));
    releaseCSRContext(handle, descr);
    printf("done\n");
}

void SubHessianCalculator::preComputeAndStoreInHost(real *hostHessianMatrix, const SvmProblem &problem,
													bool &preComputeInHost, const SVMParam &param) {
    printf("pre-compute in host\n");
    preComputeInHost = true;
    timeval start, end;
    gettimeofday(&start,NULL);
    vector<vector<KeyValue> > permutedSamples;
    for (int i = 0; i < problem.v_vSamples.size(); ++i) {
        permutedSamples.push_back(problem.v_vSamples[problem.perm[i]]);
    }
    hipsparseHandle_t handle;
    hipsparseMatDescr_t descr;
    prepareCSRContext(handle, descr);

    int m = problem.getNumOfSamples();
    int k = problem.getNumOfFeatures();
    int n = m / 100;
    real *devValA, *devValB, *devSelfDot;
    int *devRowPtrA, *devColIndA, *devRowPtrB, *devColIndB;
    real *devC;
    CSRMatrix all(permutedSamples, k);
    int nnzA = all.getNnz();
    all.copy2Dev(devValA, devRowPtrA, devColIndA);
    checkCudaErrors(hipMalloc((void **) &devSelfDot, sizeof(real) * m));
    checkCudaErrors(hipMemcpy(devSelfDot, all.getCSRValSelfDot(), sizeof(real) * m, hipMemcpyHostToDevice));
    printf("n = %d\n", n);
    float totalTime = 0;
    for (int i = 0; i < m / n + 1; ++i) {
        CSRMatrix sub(
                vector<vector<KeyValue> >(permutedSamples.begin() + n * i, permutedSamples.begin() + (n * (i + 1)>m?m:(n*(i+1)))),
                k);
        int tn = sub.getNumOfSamples();
        int nnzB = sub.getNnz();
        sub.copy2Dev(devValB, devRowPtrB, devColIndB);
        checkCudaErrors(hipMalloc((void **) &devC, sizeof(real) * tn * m));
        CSRMatrix::CSRmm2Dense(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE, tn, m, k,
                               descr, nnzB, devValB, devRowPtrB, devColIndB, descr, nnzA, devValA, devRowPtrA,
                               devColIndA, devC);
        RBFKernel << < Ceil(tn * m, BLOCK_SIZE), BLOCK_SIZE >> >
                                                (devSelfDot + n * i, devSelfDot, devC, tn, m, param.gamma);
        sub.freeDev(devValB, devRowPtrB, devColIndB);
        checkCudaErrors(
                hipMemcpy(hostHessianMatrix + n * m * i, devC, sizeof(real) * tn * m, hipMemcpyDeviceToHost));
        checkCudaErrors(hipFree(devC));
    }
    checkCudaErrors(hipFree(devSelfDot));
    releaseCSRContext(handle, descr);
    gettimeofday(&end,NULL);
    printf("time elapsed for pre-compute hessian matrix in host: %f\n", timeElapse(start,end));
}

void SubHessianCalculator::preComputeCache4BinaryProblem(real *devC, const SvmProblem &problem,
                                                         const SVMParam &param) {
    hipsparseHandle_t handle;
    hipsparseMatDescr_t descr;
    prepareCSRContext(handle, descr);
    CSRMatrix csrMatrix(problem.v_vSamples, problem.getNumOfFeatures());
    int n = problem.getNumOfSamples();
    int k = problem.getNumOfFeatures();
    computeSubHessianMatrix(handle, descr, csrMatrix, n, csrMatrix, n, k, devC, param);
    releaseCSRContext(handle, descr);
}
