/*
 * hostHessian.cpp
 *
 *  Created on: 28/10/2015
 *      Author: Zeyi Wen
 */


#include "baseHessian.h"
#include <hip/hip_runtime_api.h>
#include <sys/time.h>
#include <sys/sysinfo.h>
#include "../gpu_global_utility.h"
#include "../constant.h"
#include "../storageManager.h"

using std::endl;

//initialize the static variables for Hessian Operator
int BaseHessian::m_nTotalNumofInstance = 0;
int BaseHessian::m_nNumofDim = 0;
real* BaseHessian::m_pfHessianRowsInHostMem = NULL;
real* BaseHessian::m_pfHessianDiag = NULL;
int BaseHessian::m_nNumofCachedHessianRow = 0;

int BaseHessian::m_nNumofHessianRowsToWrite = -1;	//batch write. Group a few rows of hessian matrix to write at one time
real *BaseHessian::m_pfHessianRows = NULL;

int BaseHessian::m_nRowStartPos1 = -1;
int BaseHessian::m_nRowEndPos1 = -1;
HessianAccessor *BaseHessian::pAccessor = NULL;
FILE *BaseHessian::pHessianFile = NULL;
int BaseHessian::m_nRowStartPos2 = -1;
int BaseHessian::m_nRowEndPos2 = -1;

/*
 * @brief: set data involved in Hessian Read Operation
 * @param: nStart1: the index of the first part of a row
 * @param: nEnd1: the index of the end of the first part of a row
 * @param: nStart2: the index of the second part of a row
 * @param: nEnd2: the index of the end of the second part of a row
 */
bool BaseHessian::SetInvolveData(const int &nStart1, const int &nEnd1, const int &nStart2, const int &nEnd2)
{
	bool bReturn = false;

	if(nStart1 >= m_nTotalNumofInstance || nEnd1 >= m_nTotalNumofInstance ||
	   nStart2 >= m_nTotalNumofInstance || nEnd2 >= m_nTotalNumofInstance)
	{
		return bReturn;
	}
	m_nRowStartPos1 = nStart1;
	m_nRowEndPos1 = nEnd1;
	m_nRowStartPos2 = nStart2;
	m_nRowEndPos2 = nEnd2;

	return bReturn;
}


/*
 * @brief: allocate memory for reading content from file
 */
bool BaseHessian::AllocateBuffer(int nNumofRows)
{
	bool bReturn = false;

	if(nNumofRows < 1)
	{
		cerr << "error in hessian ops: allocate buffer failed!" << endl;
		return bReturn;
	}
	bReturn = true;
	m_pfHessianRows = new real[m_nTotalNumofInstance * nNumofRows];

	return bReturn;
}

/*
 * @brief: release buffer from reading hessian rows
 */
bool BaseHessian::ReleaseBuffer()
{
	if(m_pfHessianRows == NULL)
	{
		cerr << "buffer to be released is empty!" << endl;
		return false;
	}
	delete[] m_pfHessianRows;
	return true;
}


void BaseHessian::ReadDiagFromHessianMatrix()
{
	real *hessianRow = new real[m_nTotalNumofInstance];

	FILE *readIn = fopen(HESSIAN_FILE, "rb");
	if(readIn == NULL)
	{
		cerr << "opening file \"" << HESSIAN_FILE << "\" failed" << endl;
		exit(-1);
	}
	for(int i = 0; i < m_nTotalNumofInstance; i++)
	{
		//if the hessian row is in host memory
		if(m_nNumofCachedHessianRow > i)
		{
			long long nIndexofFirstElement = (long long) i * m_nTotalNumofInstance + i;
			m_pfHessianDiag[i] = m_pfHessianRowsInHostMem[nIndexofFirstElement];
		}
		else //the hessian row is in SSD
		{
			int nIndexInSSD = i - m_nNumofCachedHessianRow;
			ReadHessianFullRow(readIn, nIndexInSSD, 1, hessianRow);
			m_pfHessianDiag[i] = hessianRow[i];
		}
	}
	fclose(readIn);

	delete[] hessianRow;
}

bool BaseHessian::MapIndexToHessian(int &nIndex)
{
	bool bReturn = false;
	//check input parameter
	int nTempIndex = nIndex;
	if(nIndex < 0 || (nIndex > m_nRowEndPos1 && nIndex > m_nRowEndPos2))
	{
		cerr << "error in MapIndexToHessian: invalid input parameter" << endl;
		exit(0);
	}

	bReturn = true;
	if(m_nRowStartPos1 != -1)
	{
		if(nIndex <= m_nRowEndPos1)
		{
			return bReturn;
		}
		else
		{
			nTempIndex = nIndex + (m_nRowStartPos2 - m_nRowEndPos1 - 1);
			if(nTempIndex < nIndex || nTempIndex > m_nRowEndPos2)
			{
				cerr << "error in MapIndexToHessian" << endl;
				exit(0);
			}
		}
	}
	else if(m_nRowStartPos2 != -1)
	{
		nTempIndex = nIndex + m_nRowStartPos2;
		if(nTempIndex > m_nRowEndPos2)
		{
			cerr << "error in MapIndexToHessian" << endl;
			exit(0);
		}
	}
	else
	{
		cerr << "error in MapIndexToHessian: m_nStart1 & 2 equal to -1" << endl;
		exit(0);
	}

	nIndex = nTempIndex;
	return bReturn;
}

/*
 * @brief: read one full Hessian row from file
 * @return: true if read the row successfully
 */
bool BaseHessian::ReadHessianFullRow(FILE *&readIn, const int &nIndexofRow, int nNumofRowsToRead, real *pfFullHessianRow)
{
	bool bReturn = false;
	assert(readIn != NULL && nIndexofRow >= 0 && nIndexofRow < m_nTotalNumofInstance);

	//read the whole Hessian row
	bReturn = CFileOps::ReadRowsFromFile(readIn, pfFullHessianRow, m_nTotalNumofInstance, nNumofRowsToRead, nIndexofRow);
	assert(bReturn != false && pfFullHessianRow != NULL);

	return bReturn;
}

/*
 * @brief: read a continuous part of a Hessian row. Note that the last element (nEndPos) is included in the sub row
 * @output: pfHessianSubRow: part of a Hessian row
 */
bool BaseHessian::ReadHessianSubRow(FILE *&readIn, const int &nIndexofRow,
		   	   	   	   	   	   	   	  const int &nStartPos, const int &nEndPos,
		   	   	   	   	   	   	   	  real *pfHessianSubRow)
{
	bool bReturn = false;
	if(readIn == NULL || nIndexofRow < 0 || nIndexofRow > m_nTotalNumofInstance ||
		nStartPos < 0    || nEndPos < 0 	|| nStartPos > m_nTotalNumofInstance   ||
		nEndPos > m_nTotalNumofInstance)
	{
		cerr << "error in ReadHessianSubRow: invalid param" << endl;
		return bReturn;
	}

	int nNumofHessianElements = nEndPos - nStartPos + 1;//the number of elements to read

	//read the whole Hessian row
	real *pfTempFullHessianRow = new real[m_nTotalNumofInstance];
	bool bReadRow =	ReadHessianFullRow(readIn, nIndexofRow, 1, pfTempFullHessianRow); //1 means that read one Hessian row
	if(bReadRow == false)
	{
		cerr << "error in ReadHessianRow" << endl;
		delete[] pfTempFullHessianRow;
		return bReturn;
	}

	//get sub row from a full Hessian row
	memcpy(pfHessianSubRow, pfTempFullHessianRow + nStartPos, sizeof(real) * nNumofHessianElements);

	delete[] pfTempFullHessianRow;

	bReturn = true;
	return bReturn;
}

/*
 * @brief: read a few Hessian rows; This functionality is required for initialised cache, and etc. Read from nStartRow to nEndRow (include the last row)
 * @param: nNumofInvolveElements: the number of involved elements of a row
 * @param: pfHessianRow: the space to store the hessian row(s)
 * @param: nNumofElementEachRowInCache: number of element of each row in pfHessian.
 * 		   Because of the memory alignment issue, this param is usually bigger than nNumofInvolveElements
 */
bool BaseHessian::ReadHessianRows(FILE *&readIn, const int &nStartRow, const int &nEndRow,
									const int &nNumofInvolveElements, real * pfHessianRow, int nNumOfElementEachRowInCache)
{
	bool bReturn = false;
	//check input parameters
	if(readIn == NULL || nStartRow > m_nTotalNumofInstance || nEndRow > m_nTotalNumofInstance || nEndRow < nStartRow)
	{
		cerr << "error in ReadHessianRows: invalid input parameters" << endl;
		return bReturn;
	}

	//start reading Hessain sub rows
	int nSizeofFirstPart = 0;
	if(m_nRowStartPos1 != -1)
	{
		nSizeofFirstPart = m_nRowEndPos1 - m_nRowStartPos1 + 1;//the size of first part (include the last element of the part)
	}
	int nSizeofSecondPart = 0;
	if(m_nRowStartPos2 != -1)
	{
		nSizeofSecondPart = m_nRowEndPos2 - m_nRowStartPos2 + 1;
	}

	//check valid read
	if(nSizeofSecondPart + nSizeofFirstPart != nNumofInvolveElements)
	{
		cerr << "warning: reading hessian rows has potential error" << endl;
	}
	//read Hessian rows at one time
	int nNumofRows = nEndRow - nStartRow + 1;
	//float_point *pfTempHessianRows = new float_point[m_nTotalNumofSamples * nNumofRows];
	bool bReadRow =	ReadHessianFullRow(readIn, nStartRow, nNumofRows,  m_pfHessianRows);
	if(bReadRow == false)
	{
		cerr << "error in ReadHessianRow" << endl;
		return bReturn;
	}

	bReturn = true;
	//read a full Hessian row
	int nHessianEndPos;
	real *pfTempFullHessianRow;// = new float_point[m_nTotalNumofSamples];
	for(int i = nStartRow; i <= nEndRow; i++)
	{
		pfTempFullHessianRow = m_pfHessianRows + (i - nStartRow) * m_nTotalNumofInstance;

		//read the first continuous part
		if(m_nRowStartPos1 != -1)
		{
			//first part is added to the end of current Hessian space in main memory
			nHessianEndPos = (i - nStartRow) * nNumOfElementEachRowInCache;//use number of elements each row instead of number of involve elements due to memory alignment
			memcpy(pfHessianRow + nHessianEndPos, pfTempFullHessianRow + m_nRowStartPos1, sizeof(real) * nSizeofFirstPart);
		}
		//read the second continuous part
		if(m_nRowStartPos2 != -1)
		{
			nHessianEndPos = (i - nStartRow) * nNumOfElementEachRowInCache + nSizeofFirstPart;
			memcpy(pfHessianRow + nHessianEndPos, pfTempFullHessianRow + m_nRowStartPos2, sizeof(real) * nSizeofSecondPart);
		}
	}
	//delete[] pfTempHessianRows;
	return bReturn;
}

/**
 * @brief: save a pre-computed sub-matrix (rows) to host memory or SSDs
 * @pfSubMatrix: kernel values to be saved
 * @subMatrix: information about the sub-matrix
 */
void BaseHessian::SaveRows(real *pfSubMatrix, const SubMatrix &subMatrix)
{
	//store the sub matrix
	long lColStartPos = subMatrix.nColIndex;

	//the sub matrix should be stored in RAM
	int nRowId = subMatrix.nRowIndex;
	int nSubMatrixRow = subMatrix.nRowSize;
	int nSubMatrixCol = subMatrix.nColSize;

	if(nRowId + nSubMatrixRow <= m_nNumofCachedHessianRow)
	{
		//copying to host memory
		for(int k = 0; k < nSubMatrixRow; k++)
		{
			long long lPosInHessian =  (long long)(nRowId + k) * m_nTotalNumofInstance + lColStartPos;
			long lPosInSubMatrix = k * nSubMatrixCol;
			memcpy(m_pfHessianRowsInHostMem + lPosInHessian, pfSubMatrix + lPosInSubMatrix, sizeof(real) * nSubMatrixCol);
		}
	}
	else
	{
		//copy a part of the last row that can fit in host memory
		int nNumofRowsStoredInHost = 0;
		if(nRowId < m_nNumofCachedHessianRow)
		{
			nNumofRowsStoredInHost = m_nNumofCachedHessianRow - nRowId;
			//cout << "copying to host " << lColStartPos << endl;
			for(int k = 0; k < nNumofRowsStoredInHost; k++)
			{
				long long lPosInHessian =  (long long)(nRowId + k) * m_nTotalNumofInstance + lColStartPos;
				long lPosInSubMatrix = k * nSubMatrixCol;
				memcpy(m_pfHessianRowsInHostMem + lPosInHessian, pfSubMatrix + lPosInSubMatrix, sizeof(real) * nSubMatrixCol);
			}
		}

		int nNumofRowsToWrite = nSubMatrixRow - nNumofRowsStoredInHost;
		//the results of this function are: 1. write rows to file; 2. return the index of (start pos of) the rows
		long long lUnstoredStartPos =  (long long)nNumofRowsStoredInHost * nSubMatrixCol;

		//hessian sub matrix info
		SubMatrix subTempMatrix;
		subTempMatrix.nColIndex = lColStartPos;
		subTempMatrix.nColSize = nSubMatrixCol;
		subTempMatrix.nRowIndex = nRowId + nNumofRowsStoredInHost;
		//update row index in the file on ssd, as only part of the hessian matrix is stored in file
		subTempMatrix.nRowIndex -= m_nNumofCachedHessianRow;
		assert(subTempMatrix.nRowIndex >= 0);

		subTempMatrix.nRowSize = nNumofRowsToWrite;

		bool bWriteRows = pAccessor->WriteHessianRows(pHessianFile, pfSubMatrix + lUnstoredStartPos, subTempMatrix);

		if(bWriteRows == false)
		{
			cerr << "error in writing Hessian Rows" << endl;
			exit(-1);
		}
	}//end store sub matrix to file
}

/**
 * @brief: read a row from the precomputed kernel matrix
 */
void BaseHessian::ReadRow(int nPosofRowAtHessian, real *pfHessianRow)
{
	memset(pfHessianRow, 0, sizeof(real) * m_nTotalNumofInstance);
	//if the hessian row is in host memory
	if(m_nNumofCachedHessianRow > nPosofRowAtHessian)
	{
		int nSizeofFirstPart = 0;
		if(m_nRowStartPos1 != -1)
		{
			nSizeofFirstPart = m_nRowEndPos1 - m_nRowStartPos1 + 1;//the size of first part (include the last element of the part)
			long long nIndexofFirstElement = (long long)nPosofRowAtHessian * (m_nTotalNumofInstance) + m_nRowStartPos1;
			memcpy(pfHessianRow, m_pfHessianRowsInHostMem + nIndexofFirstElement, nSizeofFirstPart * sizeof(real));
		}
		if(m_nRowStartPos2 != -1)
		{
			int nSizeofSecondPart = m_nRowEndPos2 - m_nRowStartPos2 + 1;
			long long nIndexofFirstElement = (long long)nPosofRowAtHessian * (m_nTotalNumofInstance) + m_nRowStartPos2;
			memcpy(pfHessianRow + nSizeofFirstPart, m_pfHessianRowsInHostMem + nIndexofFirstElement,
				   nSizeofSecondPart * sizeof(real));
		}
	}
	else//the hessian row is in SSD
	{
		int nIndexInSSD = nPosofRowAtHessian - m_nNumofCachedHessianRow;
		pAccessor->ReadHessianRow(pHessianFile, nIndexInSSD, pfHessianRow);
	}
}


void BaseHessian::PrintHessianInfo()
{
	cout << "ins=" << m_nTotalNumofInstance << "\t";
	cout << "dim=" << m_nNumofDim << "\t";
	cout << "ram row=" << m_nNumofCachedHessianRow << "\t";
	cout << "part1_start=" << m_nRowStartPos1 << "\t";
	cout <<	"part1_end=" << m_nRowEndPos1 << "\t";
	cout <<	"part2_start=" << m_nRowStartPos2 << "\t";
	cout << "part2_end=" << m_nRowEndPos2 << endl;
}


/**
 * @brief: kernel matrix precomputation
 */
void BaseHessian::PrecomputeKernelMatrix(vector<vector<real> > &v_vDocVector, BaseHessian *hessianIOOps)
{
	//compute Hessian Matrix
	string strHessianMatrixFileName = HESSIAN_FILE;
	string strDiagHessianFileName = HESSIAN_DIAG_FILE;

	int nNumofSample = v_vDocVector.size();

	//initialize Hessian IO operator
	//CLinearKernel RBF(pfGamma[j]);

	int nNumofRowsOfHessianMatrix = v_vDocVector.size();
	//space of row-index-in-file is for improving reading performace
	BaseHessian::m_nNumofDim = v_vDocVector.front().size();
	BaseHessian::m_nTotalNumofInstance = nNumofRowsOfHessianMatrix;

	StorageManager *manager = StorageManager::getManager();
	int nNumofHessianRow = manager->RowInRAM(BaseHessian::m_nNumofDim, BaseHessian::m_nTotalNumofInstance, nNumofSample);

	cout << nNumofHessianRow << " rows cached in RAM" << endl;
	long long lSizeofCachedHessian = sizeof(real) * (long long)nNumofHessianRow * nNumofSample;


	cout << "numRow " << nNumofHessianRow << "; numIns " << nNumofSample << "; numBytes " << lSizeofCachedHessian << endl;
	if(lSizeofCachedHessian < 0)
	{
		cerr << "locate negative amount of host memory" << endl;
		exit(-1);
	}

	checkCudaErrors(hipHostMalloc((void**)&(BaseHessian::m_pfHessianRowsInHostMem), lSizeofCachedHessian));

	memset(BaseHessian::m_pfHessianRowsInHostMem, 0, lSizeofCachedHessian);
	BaseHessian::m_nNumofCachedHessianRow = nNumofHessianRow;
	BaseHessian::m_pfHessianDiag = new real[hessianIOOps->m_nTotalNumofInstance];
	//hessianIOOps->m_pfHessianDiagTest = new float_point[hessianIOOps->m_nTotalNumofInstance];

	//pre-compute Hessian Matrix and store the result into a file
	cout << "precomputing kernel matrix...";
	cout.flush();

	timeval t1, t2;
	real elapsedTime;
	gettimeofday(&t1, NULL);
	bool bWriteHessian = hessianIOOps->PrecomputeHessian(strHessianMatrixFileName, strDiagHessianFileName, v_vDocVector);
	hessianIOOps->ReadDiagFromHessianMatrix();

	gettimeofday(&t2, NULL);
	elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000.0;
	elapsedTime += (t2.tv_usec - t1.tv_usec) / 1000.0;
	//cout << "Done" << elapsedTime << " ms.\n";
	cout << " Done" << endl;

	if(bWriteHessian == false)
	{
		cerr << "write matrix to file failed" << endl;
		exit(0);
	}
}

