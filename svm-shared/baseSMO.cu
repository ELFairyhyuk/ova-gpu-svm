#include "hip/hip_runtime.h"
/*
 * baseSMO.cu
 *  @brief: definition of some sharable functions of smo solver
 *  Created on: 24 Dec 2016
 *      Author: Zeyi Wen
 */

#include "baseSMO.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include "smoGPUHelper.h"
#include "../SharedUtility/Timer.h"
#include "../SharedUtility/CudaMacro.h"

/**
 * @brief: initialise some variables of smo solver
 */
void BaseSMO::InitSolver(int nNumofTrainingIns)
{
	alpha = vector<real>(nNumofTrainingIns, 0);

    configureCudaKernel(nNumofTrainingIns);
	//allocate device memory for min/max search
	checkCudaErrors(hipMalloc((void**)&devBlockMin, sizeof(real) * numOfBlock));
	checkCudaErrors(hipMalloc((void**)&devBlockMinGlobalKey, sizeof(int) * numOfBlock));
	//for getting maximum low G value
	checkCudaErrors(hipMalloc((void**)&devBlockMinYiGValue, sizeof(real) * numOfBlock));
	checkCudaErrors(hipMalloc((void**)&devMinValue, sizeof(real)));
	checkCudaErrors(hipMalloc((void**)&devMinKey, sizeof(int)));

	checkCudaErrors(hipHostMalloc((void **) &hostBuffer, sizeof(real) * 5));
	checkCudaErrors(hipMalloc((void**)&devBuffer, sizeof(real) * 5));//only need 4 float_points

	//diagonal is frequently used in training.
	hessianDiag = new real[nNumofTrainingIns];
    checkCudaErrors(hipMalloc((void **) &devHessianDiag, sizeof(real) * nNumofTrainingIns));
}

/**
 * @brief: release solver memory
 */
void BaseSMO::DeInitSolver()
{
    checkCudaErrors(hipFree(devBlockMin));
    checkCudaErrors(hipFree(devBlockMinGlobalKey));
    checkCudaErrors(hipFree(devBlockMinYiGValue));
    checkCudaErrors(hipFree(devMinValue));
    checkCudaErrors(hipFree(devMinKey));
    checkCudaErrors(hipFree(devBuffer));
    checkCudaErrors(hipHostFree(hostBuffer));
    checkCudaErrors(hipFree(devHessianDiag));
    delete[] hessianDiag;
}

/**
 * @brief: select the first instance in SMO
 */
void BaseSMO::SelectFirst(int numTrainingInstance, real CforPositive)
{
    TIMER_START(selectTimer)
	GetBlockMinYiGValue<<<gridSize, BLOCK_SIZE>>>(devYiGValue, devAlpha, devLabel, CforPositive,
														   numTrainingInstance, devBlockMin, devBlockMinGlobalKey);
	//global reducer
	GetGlobalMin<<<1, BLOCK_SIZE>>>(devBlockMin, devBlockMinGlobalKey, numOfBlock, devYiGValue, NULL, devBuffer);

	//copy result back to host
	hipMemcpy(hostBuffer, devBuffer, sizeof(real) * 2, hipMemcpyDeviceToHost);
	IdofInstanceOne = (int)hostBuffer[0];
    TIMER_STOP(selectTimer)

	devHessianInstanceRow1 = ObtainRow(numTrainingInstance);
}

/**
 * @breif: select the second instance in SMO
 */
void BaseSMO::SelectSecond(int numTrainingInstance, real CforNegative)
{
    TIMER_START(selectTimer)
	real fUpSelfKernelValue = 0;
	fUpSelfKernelValue = hessianDiag[IdofInstanceOne];

	//for selecting the second instance
	real fMinValue;
	fMinValue = hostBuffer[1];
	upValue = -fMinValue;

	//get block level min (-b_ij*b_ij/a_ij)
	GetBlockMinLowValue<<<gridSize, BLOCK_SIZE>>>
						   (devYiGValue, devAlpha, devLabel, CforNegative, numTrainingInstance, devHessianDiag,
							devHessianInstanceRow1, upValue, fUpSelfKernelValue, devBlockMin, devBlockMinGlobalKey,
							devBlockMinYiGValue);

	//get global min
	GetGlobalMin<<<1, BLOCK_SIZE>>>
					(devBlockMin, devBlockMinGlobalKey,
					 numOfBlock, devYiGValue, devHessianInstanceRow1, devBuffer);

	//get global min YiFValue
	//0 is the size of dynamically allocated shared memory inside kernel
	GetGlobalMin<<<1, BLOCK_SIZE>>>(devBlockMinYiGValue, numOfBlock, devBuffer);

	//copy result back to host
	hipMemcpy(hostBuffer, devBuffer, sizeof(real) * 4, hipMemcpyDeviceToHost);
    TIMER_STOP(selectTimer)
}

/**
 * @brief: update two weights
 */
void BaseSMO::UpdateTwoWeight(real fMinLowValue, real fMinValue, int nHessianRowOneInMatrix,
                                     int nHessianRowTwoInMatrix, real fKernelValue, real &fY1AlphaDiff,
                                     real &fY2AlphaDiff, const int *label, real C) {
    //get YiGValue for sample one and two
    real fAlpha2 = 0;
    real fYiFValue2 = 0;
    fAlpha2 = alpha[IdofInstanceTwo];	//reserved for svm regression
    fYiFValue2 = fMinLowValue;

    //get alpha values of sample
    real fAlpha1 = 0;
    real fYiFValue1 = 0;
    fAlpha1 = alpha[IdofInstanceOne];	//reserved for svm regression
    fYiFValue1 = fMinValue;

    //Get K(x_up, x_up), and K(x_low, x_low)
    real fDiag1 = 0, fDiag2 = 0;
    fDiag1 = hessianDiag[nHessianRowOneInMatrix];
    fDiag2 = hessianDiag[nHessianRowTwoInMatrix];

    //get labels of sample one and two
    int nLabel1 = 0, nLabel2 = 0;
    nLabel1 = label[IdofInstanceOne];
    nLabel2 = label[IdofInstanceTwo];

    //compute eta
    real eta = fDiag1 + fDiag2 - 2 * fKernelValue;
    if (eta <= 0)
        eta = TAU;

    real fCost1, fCost2;
//	fCost1 = Get_C(nLabel1);
//	fCost2 = Get_C(nLabel2);
    fCost1 = fCost2 = C;

    //keep old yi*alphas
    fY1AlphaDiff = nLabel1 * fAlpha1;
    fY2AlphaDiff = nLabel2 * fAlpha2;

    //get new alpha values
    int nSign = nLabel2 * nLabel1;
    if (nSign < 0) {
        real fDelta = (-nLabel1 * fYiFValue1 - nLabel2 * fYiFValue2) / eta; //(-fYiFValue1 - fYiFValue2) / eta;
        real fAlphaDiff = fAlpha1 - fAlpha2;
        fAlpha1 += fDelta;
        fAlpha2 += fDelta;

        if (fAlphaDiff > 0) {
            if (fAlpha2 < 0) {
                fAlpha2 = 0;
                fAlpha1 = fAlphaDiff;
            }
        } else {
            if (fAlpha1 < 0) {
                fAlpha1 = 0;
                fAlpha2 = -fAlphaDiff;
            }
        }

        if (fAlphaDiff > fCost1 - fCost2) {
            if (fAlpha1 > fCost1) {
                fAlpha1 = fCost1;
                fAlpha2 = fCost1 - fAlphaDiff;
            }
        } else {
            if (fAlpha2 > fCost2) {
                fAlpha2 = fCost2;
                fAlpha1 = fCost2 + fAlphaDiff;
            }
        }
    } //end if nSign < 0
    else {
        real fDelta = (nLabel1 * fYiFValue1 - nLabel2 * fYiFValue2) / eta;
        real fSum = fAlpha1 + fAlpha2;
        fAlpha1 -= fDelta;
        fAlpha2 += fDelta;

        if (fSum > fCost1) {
            if (fAlpha1 > fCost1) {
                fAlpha1 = fCost1;
                fAlpha2 = fSum - fCost1;
            }
        } else {
            if (fAlpha2 < 0) {
                fAlpha2 = 0;
                fAlpha1 = fSum;
            }
        }
        if (fSum > fCost2) {
            if (fAlpha2 > fCost2) {
                fAlpha2 = fCost2;
                fAlpha1 = fSum - fCost2;
            }
        } else {
            if (fAlpha1 < 0) {
                fAlpha1 = 0;
                fAlpha2 = fSum;
            }
        }
    }//end get new alpha values

    alpha[IdofInstanceOne] = fAlpha1;
    alpha[IdofInstanceTwo] = fAlpha2;

    //get alpha difference
    fY1AlphaDiff = nLabel1 * fAlpha1 - fY1AlphaDiff; //(alpha1' - alpha1) * y1
    fY2AlphaDiff = nLabel2 * fAlpha2 - fY2AlphaDiff;
}

/*
 * @brief: update the optimality indicator
 */
void BaseSMO::UpdateYiGValue(int numTrainingInstance, real fY1AlphaDiff, real fY2AlphaDiff)
{
    real fAlpha1 = alpha[IdofInstanceOne];
    real fAlpha2 = alpha[IdofInstanceTwo];
    //update yiFvalue
    //copy new alpha values to device
    hostBuffer[0] = IdofInstanceOne;
    hostBuffer[1] = fAlpha1;
    hostBuffer[2] = IdofInstanceTwo;
    hostBuffer[3] = fAlpha2;
    checkCudaErrors(hipMemcpy(devBuffer, hostBuffer, sizeof(real) * 4, hipMemcpyHostToDevice));
    UpdateYiFValueKernel <<< gridSize, BLOCK_SIZE >>> (devAlpha, devBuffer, devYiGValue,
            devHessianInstanceRow1, devHessianInstanceRow2,
            fY1AlphaDiff, fY2AlphaDiff, numTrainingInstance);
    hipDeviceSynchronize();
}

void BaseSMO::configureCudaKernel(int numOfTrainingInstance) {
    //configure cuda kernel
    numOfBlock = Ceil(numOfTrainingInstance, BLOCK_SIZE);
    gridSize = dim3(numOfBlock > NUM_OF_BLOCK ? NUM_OF_BLOCK : numOfBlock, Ceil(numOfBlock, NUM_OF_BLOCK));
}

