/*
 * @author: shijiashuai
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "csrMatrix.h"

/**
 * @brief: CSR matrix constructor; construct from libsvm format data.
 */
CSRMatrix::CSRMatrix(const vector<vector<KeyValue> > &samples, int numOfFeatures) : samples(samples),
                                                                                    numOfFeatures(numOfFeatures) {
    int start = 0;
    for (int i = 0; i < samples.size(); ++i) {
        csrRowPtr.push_back(start);
        int size = samples[i].size();//get the number of features with nonzero value
        start += size;
        real sum = 0;
        for (int j = 0; j < size; ++j) {
            csrVal.push_back(samples[i][j].featureValue);
            sum += samples[i][j].featureValue * samples[i][j].featureValue;
            csrColInd.push_back(samples[i][j].id);
        }
        csrValSelfDot.push_back(sum);
    }
    csrRowPtr.push_back(start);
}

/**
 * @brief: get the number of nonzero elements of the CSR matrix.
 */
int CSRMatrix::getNnz() const {
    return csrVal.size();
}

const real *CSRMatrix::getCSRVal() const {
    return csrVal.data();
}

const real *CSRMatrix::getCSRValSelfDot() const {
    return csrValSelfDot.data();
}

const int *CSRMatrix::getCSRRowPtr() const {
    return csrRowPtr.data();
}

const int *CSRMatrix::getCSRColInd() const {
    return csrColInd.data();
}

int CSRMatrix::getNumOfSamples() const {
    return samples.size();
}

int CSRMatrix::getNumOfFeatures() const {
    return numOfFeatures;
}

/**
 * @brief: multiple two sparse matrices and output a dense matrixC.
 * @k: the dimension of training data.
 */
void CSRMatrix::CSRmm2Dense(hipsparseHandle_t handle, hipsparseOperation_t transA, hipsparseOperation_t transB, int m, int n,
                       int k, const hipsparseMatDescr_t descrA, const int nnzA, const float *valA, const int *rowPtrA,
                       const int *colIndA, const hipsparseMatDescr_t descrB, const int nnzB, const float *valB,
                       const int *rowPtrB, const int *colIndB, float *matrixC) {
    /*
     * The CSRmm2Dense result is column-major instead of row-major. To avoid transposing the result
     * we compute B'A' instead of AB' : (AB)' = B'A'
     * */
    if (transA == HIPSPARSE_OPERATION_NON_TRANSPOSE)
        transA = HIPSPARSE_OPERATION_TRANSPOSE;
    else transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    if (transB == HIPSPARSE_OPERATION_NON_TRANSPOSE)
        transB = HIPSPARSE_OPERATION_TRANSPOSE;
    else transB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    real *devA;
    checkCudaErrors(hipMalloc((void**)&devA,sizeof(real)*m*k));
    hipsparseScsr2dense(handle,m,k,descrA,valA,rowPtrA,colIndA,devA,m);
    float one(1);
    float zero(0);
    hipsparseScsrmm2(handle,transB,transA,n,m,k,nnzB,&one,descrB,valB,rowPtrB,colIndB,devA,m,&zero,matrixC,n);
    checkCudaErrors(hipFree(devA));
/**
 * the code below is csr * csr, much slower than the code above.
 */
//    if (transA == HIPSPARSE_OPERATION_NON_TRANSPOSE)
//        transA = HIPSPARSE_OPERATION_TRANSPOSE;
//    else transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
//    if (transB == HIPSPARSE_OPERATION_NON_TRANSPOSE)
//        transB = HIPSPARSE_OPERATION_TRANSPOSE;
//    else transB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
//    hipsparseMatDescr_t descrC = descrA;
//    int baseC, nnzC; // nnzTotalDevHostPtr points to host memory
//    int *nnzTotalDevHostPtr = &nnzC;
//    hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);
//    int *colIndC;
//    float *valC;
//    int *rowPtrC;
//    checkCudaErrors(hipMalloc((void **) &rowPtrC, sizeof(int) * (n + 1)));
//    hipsparseXcsrgemmNnz(handle, transB, transA, n, m, k, descrB, nnzB, rowPtrB, colIndB, descrA, nnzA, rowPtrA,
//                        colIndA, descrC, rowPtrC, nnzTotalDevHostPtr);
//    if (NULL != nnzTotalDevHostPtr) { nnzC = *nnzTotalDevHostPtr; }
//    else {
//        checkCudaErrors(hipMemcpy(&nnzC, rowPtrC + m, sizeof(int), hipMemcpyDeviceToHost));
//        checkCudaErrors(hipMemcpy(&baseC, rowPtrC, sizeof(int), hipMemcpyDeviceToHost));
//        nnzC -= baseC;
//    }
//    checkCudaErrors(hipMalloc((void **) &colIndC, sizeof(int) * nnzC));
//    checkCudaErrors(hipMalloc((void **) &valC, sizeof(float) * nnzC));
//    hipsparseScsrgemm(handle, transB, transA, n, m, k, descrB, nnzB, valB, rowPtrB, colIndB, descrA, nnzA,
//                     valA, rowPtrA, colIndA, descrC, valC, rowPtrC, colIndC);
//    hipsparseScsr2dense(handle, n, m, descrC, valC, rowPtrC, colIndC, matrixC, n);
//    checkCudaErrors(hipFree(colIndC));
//    checkCudaErrors(hipFree(valC));
//    checkCudaErrors(hipFree(rowPtrC));
}

/**
 * @brief: copy the CSR matrix to device memory.
 */
void CSRMatrix::copy2Dev(real *&devVal, int *&devRowPtr, int *&devColInd) {

    int nnz = this->getNnz();
    checkCudaErrors(hipMalloc((void **) &devVal, sizeof(real) * nnz));
    checkCudaErrors(hipMalloc((void **) &devRowPtr, sizeof(int) * (this->getNumOfSamples() + 1)));
    checkCudaErrors(hipMalloc((void **) &devColInd, sizeof(int) * nnz));
    checkCudaErrors(hipMemcpy(devVal, this->getCSRVal(), sizeof(real) * nnz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(devRowPtr, this->getCSRRowPtr(), sizeof(int) * (this->getNumOfSamples() + 1),
                               hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(devColInd, this->getCSRColInd(), sizeof(int) * nnz, hipMemcpyHostToDevice));
}
void CSRMatrix::copy2Dev(real *&devVal, int *&devRowPtr, int *&devColInd, real *&devSelfDot) {
    this->copy2Dev(devVal, devRowPtr, devColInd);
    checkCudaErrors(hipMalloc((void **) &devSelfDot, sizeof(int) * getNumOfSamples()));
    checkCudaErrors(hipMemcpy(devSelfDot, this->getCSRValSelfDot(), sizeof(int) * getNumOfSamples(), hipMemcpyHostToDevice));
}
/**
 * @brief: release the device CSR matrix
 */
void CSRMatrix::freeDev(real *&devVal, int *&devRowPtr, int *&devColInd) {
    checkCudaErrors(hipFree(devVal));
    checkCudaErrors(hipFree(devRowPtr));
    checkCudaErrors(hipFree(devColInd));
}

void CSRMatrix::freeDev(real *&devVal, int *&devRowPtr, int *&devColInd, real *&devSelfDot) {
    checkCudaErrors(hipFree(devVal));
    checkCudaErrors(hipFree(devRowPtr));
    checkCudaErrors(hipFree(devColInd));
    checkCudaErrors(hipFree(devSelfDot));
}
