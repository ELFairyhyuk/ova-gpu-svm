#include "hip/hip_runtime.h"
/**
 * LinearCalculater.cu
 * Created on: May 14, 2013
 * Author: Zeyi Wen
 * Copyright @DBGroup University of Melbourne
 **/

#include "kernelCalculater.h"
#include "kernelCalGPUHelper.h"
#include "../my_assert.h"

/*
 * @brief: compute a certain # of rows of the Hessian Matrix by RBF function
 * @param: pfDevSamples: a device pointer to the whole samples. These samples indicate which rows are computed in this round
 * @param: pfDevTransSamples: a device pointer to the whole samples with transposition
 * @param: pfdevHessianRows: a device pointer to a certain # of Hessian Matrix rows to be computed
 * @param: nNumofSamples: indicates the length of pfDevTransSamples
 * @param: nNumofRows: indicates the length of pfDevSamples
 */
bool CLinearKernel::ComputeHessianRows(real *pfDevSamples, real *pfDevTransSamples, real *pfDevHessianRows,
									const int &nNumofSamples, const int &nNumofDim,
									const int &nNumofRows, const int &nStartRow)
{
	bool bReturn = true;

	int nBlockSize = 0;
	dim3 dimGrid;
	GetGPUSpec(dimGrid, nBlockSize, nNumofSamples, nNumofRows);
	assert(nBlockSize >= 0);
	LinearKernel<<<dimGrid, nBlockSize, nBlockSize * sizeof(real)>>>
				(pfDevSamples, pfDevTransSamples, pfDevHessianRows, nNumofSamples, nNumofDim, nStartRow);
	hipDeviceSynchronize();
	assert(hipGetLastError() == hipSuccess);

	return bReturn;
}
