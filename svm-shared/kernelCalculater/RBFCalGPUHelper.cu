#include "hip/hip_runtime.h"

#include "kernelCalGPUHelper.h"
/*
 * @brief: compute one Hessian row
 * @param: pfDevSamples: data of samples. One dimension array represents a matrix
 * @param: pfDevTransSamples: transpose data of samples
 * @param: pfDevHessianRows: a Hessian row. the final result of this function
 * @param: nNumofSamples: the number of samples
 * @param: nNumofDim: the number of dimensions for samples
 * @param: nStartRow: the Hessian row to be computed
 */
__device__ void RBFOneRow(real *pfDevSamples, real *pfDevTransSamples,
						  real *pfDevHessianRows, int nNumofSamples, int nNumofDim,
						  int nStartRow, real fGamma)
{
	int nThreadId = threadIdx.x;
	int nBlockSize = blockDim.x;
	int nGlobalIndex = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;//global index for thread
	extern __shared__ real fSampleValue[];

	int nTempPos = 0;
	real fKernelValue = 0;
	int nRemainDim = nNumofDim;

	if(nThreadId >= nBlockSize)
	{
		return;
	}

	//when the # of dimension is huge, we process a part of dimension. one thread per kernel value
	for(int j = 0; nRemainDim > 0; j++)
	{
		//if(nThreadId == 0)
		//{
		//starting position of a sample, + j * nNumofValuesInShdMem is for case that dimension is too large
		nTempPos = nStartRow * nNumofDim + j * nBlockSize;
		//}
		//__syncthreads();

		//in the case, nThreadId < nMaxNumofThreads
		//load (part of or all of) the sample values into shared memory
		if(nThreadId < nRemainDim)
		{
			fSampleValue[nThreadId] = pfDevSamples[nTempPos + nThreadId];
		}
		__syncthreads(); //synchronize threads within a block

		/* start compute kernel value */
		if(nGlobalIndex < nNumofSamples)
		{
			real fTempSampleValue;
			real fDiff;
			//when the block size is larger than remaining dim, k is bounded by nRemainDim
			//when the nRemainDim is larger than block size, k is bounded by nBlockSize
			for(int k = 0; (k < nBlockSize) && (k < nRemainDim); k++)
			{
				nTempPos = (nNumofDim - nRemainDim + k) * nNumofSamples + nGlobalIndex;
				fTempSampleValue = pfDevTransSamples[nTempPos]; //transpose sample
				fDiff = fSampleValue[k] - fTempSampleValue;

				fKernelValue += (fDiff * fDiff);
			}
		}

		nRemainDim -= nBlockSize;
		//synchronize threads within block to avoid modifying shared memory
		__syncthreads();
	}//end computing one kernel value

	//load the element to global
	if(nGlobalIndex < nNumofSamples)
	{
		fKernelValue = fKernelValue * fGamma;
		fKernelValue = -fKernelValue; //Gaussian kernel use "-gamma"
		if(sizeof(real) == sizeof(double))
			fKernelValue = exp(fKernelValue);
		else
			fKernelValue = expf(fKernelValue);

		pfDevHessianRows[nGlobalIndex] = fKernelValue;
	}
}


//a few blocks compute one row of the Hessian matrix. The # of threads invovled in a row is equal to the # of samples
//one thread an element of the row
//the # of thread is equal to the # of dimensions or the available size of shared memory
__global__ void RBFKernel(real *pfDevSamples, real *pfDevTransSamples, real *pfDevHessianRows,
						  int nNumofSamples, int nNumofDim, int nNumofRows, int nStartRow, real fGamma)
{
	real *pfDevTempHessianRow;

	//for(int i = 0; i < nNumofRows; i++)
	{
		//pointer to a hessian row
		//pfDevTempHessianRow = pfDevHessianRows + i * nNumofSamples;
		pfDevTempHessianRow = pfDevHessianRows + blockIdx.z * nNumofSamples;

		RBFOneRow(pfDevSamples, pfDevTransSamples, pfDevTempHessianRow,
				  nNumofSamples, nNumofDim, nStartRow + blockIdx.z, fGamma);
		//nStartRow++;//increase to next row
	} //end computing n rows of Hessian Matrix

}

//a few blocks compute one row of the Hessian matrix. The # of threads invovled in a row is equal to the # of samples
//one thread an element of the row
//the # of thread is equal to the # of dimensions or the available size of shared memory
__global__ void ObtainRBFKernel(real *pfDevHessianRows, real *pfDevSelfDot, int nNumofSamples,
								int nNumofRows, real fGamma, int nStartRow, int nStartCol)
{
	int nGlobalIndex = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;//global index for thread
	if(nGlobalIndex < nNumofSamples * nNumofRows)
	{
		int nRow = (nGlobalIndex / nNumofSamples + nStartRow);
		int nCol = (nGlobalIndex % nNumofSamples + nStartCol);


		float fKernelValue = (pfDevSelfDot[nRow] + pfDevSelfDot[nCol] -pfDevHessianRows[nGlobalIndex] * 2.f ) * fGamma;
		fKernelValue = -fKernelValue; //Gaussian kernel use "-gamma"
		if(sizeof(real) == sizeof(double))
			fKernelValue = exp(fKernelValue);
		else
			fKernelValue = expf(fKernelValue);

		//if(nGlobalIndex == 299 * nNumofSamples + 100)
		//	printf("%f, %f, %f, %f, %d\n", pfDevSelfDot[nCol], pfDevSelfDot[nRow], pfDevHessianRows[nGlobalIndex], fKernelValue, nGlobalIndex);

		pfDevHessianRows[nGlobalIndex] = fKernelValue;
	}
}

//a few blocks compute one row of the Hessian matrix. The # of threads involved in a row is equal to the # of samples
//one thread an element of the row
//the # of thread is equal to the # of dimensions or the available size of shared memory
__global__ void UpdateDiag(real *pfDevHessianRows, int nNumofSamples, int nNumofRows)
{
	int nGlobalIndex = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;//global index for thread
	if(nGlobalIndex < nNumofSamples * nNumofRows)
	{
		int nRow = nGlobalIndex / nNumofSamples;
		int nCol = nGlobalIndex % nNumofSamples;
		if(nRow == nCol)
			pfDevHessianRows[nGlobalIndex] = 1;
	}
}
