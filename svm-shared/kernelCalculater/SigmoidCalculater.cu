#include "hip/hip_runtime.h"
/*
 * SigmoidCalculater.cu
 *
 * Created on: 28/05/2013
 * Author: Zeyi Wen
 * Copyright @DBGroup University of Melbourne
 **/


#include "kernelCalculater.h"
#include "kernelCalGPUHelper.h"
#include "../my_assert.h"

/*
 * @brief: compute a certain # of rows of the Hessian Matrix by Polynomial function
 * @param: pfDevSamples: a device pointer to the whole samples. These samples indicate which rows are computed in this round
 * @param: pfDevTransSamples: a device pointer to the whole samples with transposition
 * @param: pfdevHessianRows: a device pointer to a certain # of Hessian Matrix rows to be computed
 * @param: nNumofSamples: indicates the length of pfDevTransSamples
 * @param: nNumofRows: indicates the length of pfDevSamples
 */
bool CSigmoidKernel::ComputeHessianRows(real *pfDevSamples, real *pfDevTransSamples, real *pfDevHessianRows,
									const int &nNumofSamples, const int &nNumofDim,
									const int &nNumofRows, const int &nStartRow)
{
	bool bReturn = true;

	int nBlockSize = 0;
	dim3 dimGrid;
	GetGPUSpec(dimGrid, nBlockSize, nNumofSamples, nNumofRows);
	assert(nBlockSize >= 0);
	SigmoidKernel<<<dimGrid, nBlockSize, nBlockSize * sizeof(real)>>>(pfDevSamples,
			pfDevTransSamples, pfDevHessianRows, nNumofSamples, nNumofDim, nStartRow, m_fCoef);

	hipDeviceSynchronize();
	assert(hipGetLastError() == hipSuccess);

	return bReturn;
}



