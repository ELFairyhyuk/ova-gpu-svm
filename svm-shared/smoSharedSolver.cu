#include "hip/hip_runtime.h"

#include "smoSolver.h"
#include "smoGPUHelper.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
//#include <cutil.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
#include <sys/time.h>

/*
 * @brief: allocate GPU memory for finding block min, which serves as the first sample of the optimized pair
 * @param: nNumofTrainingSamples: the # of training samples in current interation
 * @return: true when memory allocation success
 */
bool CSMOSolver::SMOSolverPreparation(const int &nNumofTrainingSamples)
{
	bool bReturn = true;
	InitSolver(nNumofTrainingSamples);

	//allocate memory in CPU
	//m_pfHessianRow = new float_point[nNumofTrainingSamples];//for reading hessian row from file
	hipHostMalloc(&m_pfHessianRow, sizeof(real) * nNumofTrainingSamples);
	m_pnLabel = new int[nNumofTrainingSamples];

	return bReturn;
}


/*
 * @brief: release memory used for caching
 */
bool CSMOSolver::CleanCache()
{
	bool bReturn = true;

	//clean cache
	m_pGPUCache->CleanCache();
	checkCudaErrors(hipFree(m_pfDevHessianMatrixCache));

	if(hipGetLastError() != hipSuccess)
	{
		cerr << "CUDA error occurs at CleanCache" << endl;
		bReturn = false;
	}

	return bReturn;
}

/*
 * @brief: release memory used by SMO slover
 */
bool CSMOSolver::SMOSolverEnd()
{
	bool bReturn = true;

	//free GPU global memory
	DeInitSolver();

	hipHostFree(m_pfHessianRow);
	delete[] m_pnLabel;
	delete[] m_pfGValue;

	return bReturn;
}

/*
 * @brief: set data used by SMO solver in Hessian Matrix
 */
bool CSMOSolver::SetInvolveData(int nStart1, int nEnd1, int nStart2, int nEnd2)
{
	bool bReturn = true;

	m_nStart1 = nStart1;
	m_nEnd1 = nEnd1;
	m_nStart2 = nStart2;
	m_nEnd2 = nEnd2;

	bReturn = m_pHessianReader->SetInvolveData(nStart1, nEnd1, nStart2, nEnd2);

	return bReturn;
}

/*
 * @brief: map a given index to Hessian matrix. As in n-fold-cross-validation, data are seperated into n parts.
 */
bool CSMOSolver::MapIndexToHessian(int &nIndex)
{
	bool bReturn = false;
	//check input parameter
	int nTempIndex = nIndex;
	if(nIndex < 0 || (nIndex > m_nEnd1 && nIndex > m_nEnd2))
	{
		cerr << "error in MapIndexToHessian: invalid input parameter" << endl;
		cout << nIndex << " " << m_nEnd1 << " " << m_nEnd2 << endl;
		exit(0);
	}

	bReturn = true;
	if(m_nStart1 != -1)
	{
		if(nIndex <= m_nEnd1)
		{
			return bReturn;
		}
		else
		{
			nTempIndex = nIndex + (m_nStart2 - m_nEnd1 - 1);
			if(nTempIndex < nIndex || nTempIndex > m_nEnd2)
			{
				cerr << "error in MapIndexToHessian" << endl;
				bReturn = false;
			}
		}
	}
	else if(m_nStart2 != -1)
	{
		nTempIndex = nIndex + m_nStart2;
		if(nTempIndex > m_nEnd2)
		{
			cerr << "error in MapIndexToHessian" << endl;
			bReturn = false;
		}
	}
	else
	{
		cerr << "error in MapIndexToHessian: m_nStart1 & 2 equal to -1" << endl;
		bReturn = false;
	}

	nIndex = nTempIndex;
	return bReturn;
}


/*
 * @brief: get a row of Hessian Matrix (the row is either in cache or in secondary memory)
 */
long lGetHessianRowTime = 0;
long readRowTime = 0;
long cacheMissMemcpyTime = 0;
long lGetHessianRowCounter = 0;
long cacheMissCount = 0;
long lRamHitCount = 0;
long lSSDHitCount = 0;
real* CSMOSolver::GetHessianRow(const int &nNumofInstance, const int &nPosofRow)
{
	timespec time1, time2, time3, time4, time5;
	clock_gettime(CLOCK_REALTIME, &time1);
	lGetHessianRowCounter++;
    /*printf("get row %d label %d\n",problem->originalIndex[nPosofRow], problem->originalLabel[nPosofRow]);*/
//	assert(nNumofInstance >= nPosofRow);

	real *pfDevHessianRow = NULL;
	//get 1st row
	int nCacheLocation = -1;
	bool bIsCacheFull = false;
	bool bIsInCache = m_pGPUCache->GetDataFromCache(nPosofRow, nCacheLocation, bIsCacheFull);

	long long lCachePosStart = (long long)nCacheLocation * m_lNumofElementEachRowInCache;

	if(bIsInCache == false)
	{//cache missed
		clock_gettime(CLOCK_REALTIME, &time3);
		if(bIsCacheFull == true)
			m_pGPUCache->ReplaceExpired(nPosofRow, nCacheLocation, NULL);
		//convert current training position to the position in Hessian matrix
		int nPosofRowAtHessian = nPosofRow;
		bool bMapIndex = MapIndexToHessian(nPosofRowAtHessian);
		assert(bMapIndex == true);

//		memset(m_pfHessianRow, 0, sizeof(float_point) * nNumofInstance);
		m_pHessianReader->ReadRow(nPosofRowAtHessian, m_pfHessianRow);

		clock_gettime(CLOCK_REALTIME, &time4);

//		cout << nCacheLocation << "; cache is full=" << bIsCacheFull << endl;
		lCachePosStart = (long long)nCacheLocation * m_lNumofElementEachRowInCache;
		//checkCudaErrors(hipMemcpyAsync(m_pfDevHessianMatrixCache + lCachePosStart, m_pfHessianRow,
		//					  	  		sizeof(float_point) * nNumofInstance, hipMemcpyHostToDevice, m_stream1_Hessian_row));
		checkCudaErrors(hipMemcpy((m_pfDevHessianMatrixCache + lCachePosStart), m_pfHessianRow, sizeof(real) * nNumofInstance, hipMemcpyHostToDevice));

		cacheMissCount++;
		clock_gettime(CLOCK_REALTIME, &time5);
		long lTemp = ((time5.tv_sec - time3.tv_sec) * 1e9 + (time5.tv_nsec - time3.tv_nsec));
		readRowTime += lTemp;
		lTemp = ((time5.tv_sec - time4.tv_sec) * 1e9 + (time5.tv_nsec - time4.tv_nsec));
        cacheMissMemcpyTime += lTemp;
	}

	pfDevHessianRow = m_pfDevHessianMatrixCache + lCachePosStart;
	clock_gettime(CLOCK_REALTIME, &time2);
	long lTemp = ((time2.tv_sec - time1.tv_sec) * 1e9 + (time2.tv_nsec - time1.tv_nsec));
	if(lTemp > 0)
	{
		lGetHessianRowTime += lTemp;
	}

	return pfDevHessianRow;
}

